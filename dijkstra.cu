
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

#define DEBUG 0

__global__ void find_minimum_kernel(int *array, int *min, int *mutex, unsigned int n) {
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int stride = gridDim.x * blockDim.x;
	unsigned int offset = 0;
	int temp_min = 1000;

	__shared__ int cache[256];

	while(index + offset < n) {
		temp_min = fminf(temp_min, array[index + offset]);
		offset += stride;
	}

	cache[threadIdx.x] = temp_min;

	__syncthreads();

	unsigned int iii = blockDim.x / 2;
	while(iii > 0) {
		if(threadIdx.x < iii) {
			cache[threadIdx.x] = fminf(cache[threadIdx.x], cache[threadIdx.x + iii]);
		}
		__syncthreads();
		iii >>= 1;
	}

	if(threadIdx.x == 0) {
		while(atomicCAS(mutex, 0, 1) != 0); // acquire mutex
		*min = fminf(*min, cache[0]);
		atomicExch(mutex, 0); // unlock mutex
	}
}

void dijktra(int **graph, int size, int src) {
	for(int iii = 0; iii < size; iii++) {
		for(int jjj = 0; jjj < size; jjj++) {
			printf("%d ", graph[iii][jjj]);
		}
		printf("\n");
	}
}

int read_file(char *file_name, int *vertices, int **incidence_matrix) {
	FILE *f;
	int v;

	f = fopen(file_name, "r");
	if(f == NULL) {
		return 1;
	}

	fscanf(f, "%d\n", vertices);
	v = *vertices; // this is meant for readability later, not any optimizations
	*incidence_matrix = (int*) malloc(sizeof(int) * v * v);
	printf("matrix made\n");
	for(int iii = 0; iii < v; iii++) {
		printf("iii = %d\n", iii);
		for(int jjj = 0; jjj < v; jjj++) {
			if(!fscanf(f, "%d", &incidence_matrix[iii][jjj])) break;
			else printf("%d\n", incidence_matrix[iii][jjj]);
		}
	}

	return 0;
}

int main(int argc, char *argv[]) {
	int *incidence_matrix;
	int vertices;

	if(argc != 3) {
		printf("Incorrect usage\n"); // sanity check
		printf("Correct usage: ./dijktra path_to_file source_vertex");
		exit(1);
	}
	
	if(read_file(argv[1], &vertices, &incidence_matrix)) {
		printf("Error reading file\n");
		exit(1);
	}

	dijktra((int**) incidence_matrix, vertices, atoi(argv[2]));

	return 0;
}
