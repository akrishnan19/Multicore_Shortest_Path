
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <iostream>
#include <limits.h> 
#include <malloc.h>
#include <omp.h>

__global__ void fords_kernel(int n, int u, int *mat, int *dist) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  if(index < n) {
    for(int v = index; v < n; v += stride) {
      int weight = mat[u * n + v];
      if(weight) {
        int temp = dist[u] + weight;
        if(temp < dist[v]) {
          dist[v] = temp;
        }
      }
    }
  }
}

void printResults(int dist[], int n, int source) { 
  printf("Vertex\t\tDistance from Source Vertex %d\n", source);
  for (int i = 0; i < n; i++)
    printf("%d\t\t%d\n", i, dist[i]);
  printf("\n\n");
} 

void fords(int *incidence_matrix, int n, int src) {
  int *h_dist;
  int *d_mat, *d_dist;

  h_dist = (int*) malloc(sizeof(int) * n);
  hipMalloc((void**) &d_mat, sizeof(int) * n *n);
  hipMalloc((void**) &d_dist, sizeof(int) * n);

  #pragma omp parallel for
  for(int i = 0 ; i < n; i ++){
    h_dist[i] = INT_MAX;
  }

  h_dist[src] = 0;

  hipMemcpy(d_mat, incidence_matrix, sizeof(int) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(d_dist, h_dist, sizeof(int) * n, hipMemcpyHostToDevice);

  dim3 THREAD_SIZE = 256; // can't use variable size so everything is hard-coded
  dim3 BLOCK_SIZE = 256;

  for(int iii = 0; iii < n; iii++) {
    fords_kernel<<< BLOCK_SIZE, THREAD_SIZE >>>(n, iii, d_mat, d_dist);
  }
  
  hipMemcpy(h_dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);

  printResults(h_dist, n, src);

  hipFree(d_mat);
  hipFree(d_dist);
}

int* read_file(char *file_name, int *vertices) {
  FILE *f;
  int *incidence_matrix;
  int v;

  f = fopen(file_name, "r");
  if(f == NULL) {
    printf("Error reading file\n");
    exit(1);
  }

  fscanf(f, "%d\n", vertices);
  
  v = *vertices; // this is meant for readability later, not any optimizations
  incidence_matrix = (int*) malloc(sizeof(int) * v * v);

  for(int iii = 0; iii < v; iii++) {
    for(int jjj = 0; jjj < v; jjj++) {
      fscanf(f, "%d", &incidence_matrix[iii * v + jjj]);
      if(incidence_matrix[iii* v + jjj] < 0) {
        printf("Error: negative edge weight found at %d, %d\n", iii, jjj);
        exit(-1);
      }
    }
  }

  return incidence_matrix;
}

int main(int argc, char *argv[]) {
    int *incidence_matrix;
    int vertices;

    if(argc != 2) {
        printf("Incorrect usage\n"); // sanity check
        printf("Correct usage: ./ford path_to_file\n");
        exit(1);
    }
    
    incidence_matrix = read_file(argv[1], &vertices);

    for(int iii = 0; iii < vertices; iii++)
      fords(incidence_matrix, vertices, iii);

    free(incidence_matrix);

    return 0;
}
